#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/onezero_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void OnezeroForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    if(in[index]>0.001)
        out[index] = 1;
    else
        out[index] = in[index] > 0 ? 1000*in[index] : 0;
  }
}

template <typename Dtype>
void OnezeroLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  OnezeroForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;

}

template <typename Dtype>
__global__ void OnezeroBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    if(in_data[index]>0.001)
        out_diff[index] = 0;
    else
        out_diff[index] = in_data[index] > 0 ? 1000*in_diff[index] : 0;
  }
}

template <typename Dtype>
void OnezeroLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    OnezeroBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(OnezeroLayer);


}  // namespace caffe
