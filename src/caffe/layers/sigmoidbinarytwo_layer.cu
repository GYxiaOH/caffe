#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>

#include "caffe/layers/sigmoidbinarytwo_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SigmoidbinarytwoForward(const int n, const Dtype* in, Dtype* out,Dtype a) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = 2. / (1. + exp(-2*a*in[index])) - 1;
  }
}

template <typename Dtype>
void SigmoidbinarytwoLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  Dtype t = this->layer_param_.sigmoidbinarytwo_param().t();
  Dtype e = this->layer_param_.sigmoidbinarytwo_param().e();
  Dtype a = log((2-t)/t)/e;
  SigmoidbinarytwoForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data,a);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void SigmoidbinarytwoBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, Dtype* out_diff,Dtype a) {
  CUDA_KERNEL_LOOP(index, n) {
    const Dtype sigmoid_x = (out_data[index]+1) * 0.5;
    out_diff[index] = in_diff[index] * sigmoid_x * (1 - sigmoid_x)*a;
  }
}

template <typename Dtype>
void SigmoidbinarytwoLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    Dtype t = this->layer_param_.sigmoidbinarytwo_param().t();
    Dtype e = this->layer_param_.sigmoidbinarytwo_param().e();
    Dtype a = log((2-t)/t)/e;
    // NOLINT_NEXT_LINE(whitespace/operators)
    SigmoidbinarytwoBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top_data, bottom_diff,a);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SigmoidbinarytwoLayer);


}  // namespace caffe
