#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>

#include "caffe/layers/sigmoidnew_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SigmoidnewForward(const int n, const Dtype* in, Dtype* out,Dtype a,Dtype e) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = 1. / (1. + exp(-a*(in[index]-(e/2))));
  }
}

template <typename Dtype>
void SigmoidnewLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  Dtype t = this->layer_param_.sigmodnew_param().t();
  Dtype e = this->layer_param_.sigmodnew_param().e();
  Dtype a = 2*log((1-t)/t)/e;
  SigmoidnewForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data,a,e);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void SigmoidnewBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, Dtype* out_diff,Dtype a) {
  CUDA_KERNEL_LOOP(index, n) {
    const Dtype sigmoid_x = out_data[index];
    out_diff[index] = in_diff[index] * sigmoid_x * (1 - sigmoid_x)*a;
  }
}

template <typename Dtype>
void SigmoidnewLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    Dtype t = this->layer_param_.sigmodnew_param().t();
    Dtype e = this->layer_param_.sigmodnew_param().e();
    Dtype a = 2*log((1-t)/t)/e;
    // NOLINT_NEXT_LINE(whitespace/operators)
    SigmoidnewBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top_data, bottom_diff,a);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SigmoidnewLayer);


}  // namespace caffe
